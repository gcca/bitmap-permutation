#include "hip/hip_runtime.h"
#include <gtest/gtest.h>

#include <cstdint>

__device__ std::uint8_t bitmask(const std::uint8_t *bitmap,
                                const std::size_t bitmap_offset)
{
  const std::size_t bitmap_index = bitmap_offset / 8;
  const std::size_t bit_offset = bitmap_offset % 8;

  return bitmap[bitmap_index] & (128 >> bit_offset) ? 1 << bit_offset : 0;
}


__global__ void bitmap_permutation(const std::uint8_t *bitmap,
                                   const std::size_t *map,
                                   const std::size_t map_size,
                                   std::uint8_t *permutation)
{
  const std::size_t chunk_start = (blockIdx.x * blockDim.x + threadIdx.x) * 8;

  std::uint8_t result = 0;

  for (std::size_t i = 0; i < 8; i++)
    {
      result ^= bitmask(bitmap, map[chunk_start + i]);
    }

  permutation[chunk_start] = result;
}


TEST(bitmapPermutationTest, FirstAttempt)
{
  std::uint8_t *bitmap;  // TODO: (int) type can be a parameter
  std::uint8_t *permutation;
  hipMallocManaged(&bitmap, 2 * sizeof(std::uint8_t));
  hipMallocManaged(&permutation, 2 * sizeof(std::uint8_t));

  bitmap[0] = 0b01010101;
  bitmap[1] = 0b01010101;

  permutation[0] = 0;
  permutation[1] = 0;


  std::size_t *map;  // TODO: type as parameter for indexing
  hipMallocManaged(&map, 10 * sizeof(std::size_t));

  const std::size_t input_map[] = {7, 6, 5, 4, 3, 2, 1, 0, 15, 14};

  for (std::size_t i = 0; i < 10; i++)
    {
      map[i] = input_map[i];
    }

  // TODO: threads multiple of 8
  bitmap_permutation<<<1, 2>>>(bitmap, map, 10, permutation);

  hipDeviceSynchronize();

  EXPECT_EQ(0b10101010, permutation[0]);
  EXPECT_EQ(0b10000000, permutation[1]);

  hipFree(bitmap);
  hipFree(permutation);

  hipFree(map);
}
